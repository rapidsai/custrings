#include <memory>
#include <string.h>
#include <stdlib.h>
#include <string>
#include <vector>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/for_each.h>
#include "../include/NVStrings.h"

//
// cd ../build
// nvcc -w -std=c++11 --expt-extended-lambda -gencode arch=compute_70,code=sm_70 ../tests/csv.cu -L. -lNVStrings -o csv --linker-options -rpath,.:
//

// csv file contents in device memory
void* d_fileContents = 0;

// return a vector of DString's we wish to process
std::pair<const char*,size_t>* setupTest(int& linesCount, int column)
{
    //FILE* fp = fopen("../../data/1420-rows.csv", "rb");
    //FILE* fp = fopen("../../data/7584-rows.csv", "rb");
    FILE* fp = fopen("../../data/36634-rows.csv", "rb");
    if( !fp )
    {
        printf("missing csv file\n");
        return 0;
    }
	fseek(fp, 0, SEEK_END);
	int fileSize = (int)ftell(fp);
    fseek(fp, 0, SEEK_SET);
    printf("File size = %d bytes\n", fileSize);
    if( fileSize < 2 )
    {
        fclose(fp);
        return 0;
    }
    // load file into memory
    int contentsSize = fileSize+2;
    char* contents = new char[contentsSize+2];
    fread(contents, 1, fileSize, fp);
    contents[fileSize] = '\r'; // line terminate
	contents[fileSize+1] = 0;  // and null-terminate
	fclose(fp);

    // find lines -- compute offsets vector values
    thrust::host_vector<int> lineOffsets;
    char* ptr = contents;
    while( *ptr )
    {
        char ch = *ptr;
        if( ch=='\r' )
        {
            *ptr = 0;
            while(ch && (ch < ' ')) ch = *(++ptr);
            lineOffsets.push_back((int)(ptr - contents));
            continue;
        }    
        ++ptr;
    }
    linesCount = (int)lineOffsets.size();
    printf("Found %d lines\n",linesCount);
    // copy file contents into device memory
    char* d_contents = 0;
    hipMalloc(&d_contents,contentsSize);
    hipMemcpy(d_contents,contents,contentsSize,hipMemcpyHostToDevice);
    delete contents; // done with the host data

    // copy offsets vector into device memory
    thrust::device_vector<int> offsets(lineOffsets);
    int* d_offsets = offsets.data().get();
    // build empty output vector of DString*'s
    --linesCount; // removed header line
    std::pair<const char*,size_t>* d_column1 = 0;
    hipMalloc(&d_column1, linesCount * sizeof(std::pair<const char*,size_t>));

    // create a vector of DStrings using the first column of each line
    thrust::for_each_n(thrust::device,
      thrust::make_counting_iterator<size_t>(0), linesCount, 
      [d_contents, d_offsets, column, d_column1] __device__(size_t idx){
        // probably some more elegant way to do this
        int lineOffset = d_offsets[idx];
        int lineLength = d_offsets[idx+1] - lineOffset;
        d_column1[idx].first = (const char*)0;
        if( lineLength < 1 )
            return;
        char* line = &(d_contents[lineOffset]);
        char* stringStart = line;
        int columnLength = 0, col = 0;
        for( int i=0; (i < lineLength); ++i )
        {
            if( line[i] && line[i] != ',' )
            {
                ++columnLength;
                continue;
            }
            if( col++ >= column )
                break;
            stringStart = line + i + 1;
            columnLength = 0;
        }
        if( columnLength==0 )
            return;
        // add string to vector array
        d_column1[idx].first = (const char*)stringStart;
        d_column1[idx].second = (size_t)columnLength;
      });
    //
    hipDeviceSynchronize();
    d_fileContents = d_contents;
    return d_column1;
}

int main( int argc, char** argv )
{
    //NVStrings::initLibrary();

    int count = 0;
    std::pair<const char*,size_t>* column1 = setupTest(count,2);
    if( column1==0 )
        return -1;

    NVStrings* dstrs = NVStrings::create_from_index( column1, count );

    hipFree(d_fileContents); // csv data not needed once dstrs is created
    hipFree(column1);        // string index data has done its job as well

    // simple strings op
    int* rtn = new int[count];
    dstrs->len(rtn,false);
    for( int idx=0; idx < count; ++idx )
        printf("%d,",rtn[idx]);
    printf("\n");
    delete rtn;

    // show column values
    dstrs->print();

    NVStrings::destroy(dstrs);
    return 0;
}