
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/for_each.h>
#include <rmm/rmm.h>
#include <rmm/thrust_rmm_allocator.h>
#include "NVStrings.h"
#include "NVStringsImpl.h"
#include "custring_view.cuh"
#include "Timing.h"


// remove the target characters from the beginning of each string
NVStrings* NVStrings::lstrip( const char* to_strip )
{
    unsigned int count = size();
    custring_view_array d_strings = pImpl->getStringsPtr();
    auto execpol = rmm::exec_policy(0);

    char* d_strip = 0;
    if( to_strip )
    {
        int len = (int)strlen(to_strip) + 1; // include null
        RMM_ALLOC(&d_strip,len,0);
        hipMemcpy(d_strip,to_strip,len,hipMemcpyHostToDevice);
    }

    // compute size of output buffer
    rmm::device_vector<size_t> lengths(count,0);
    size_t* d_lengths = lengths.data().get();
    double st1 = GetTime();
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
        [d_strings, d_strip, d_lengths] __device__(unsigned int idx){
            custring_view* dstr = d_strings[idx];
            if( !dstr )
                return;
            unsigned int len = dstr->lstrip_size(d_strip);
            len = ALIGN_SIZE(len);
            d_lengths[idx] = (size_t)len;
        });
    // create output object
    NVStrings* rtn = new NVStrings(count);
    char* d_buffer = rtn->pImpl->createMemoryFor(d_lengths);
    if( d_buffer==0 )
    {
        if( d_strip )
            RMM_FREE(d_strip,0);
        return rtn; // all strings are null
    }

    double et1 = GetTime();
    // create offsets
    rmm::device_vector<size_t> offsets(count,0);
    thrust::exclusive_scan(execpol->on(0),lengths.begin(),lengths.end(),offsets.begin());
    // do the strip
    custring_view** d_results = rtn->pImpl->getStringsPtr();
    size_t* d_offsets = offsets.data().get();
    double st2 = GetTime();
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
        [d_strings, d_strip, d_buffer, d_offsets, d_results] __device__(unsigned int idx){
            custring_view* dstr = d_strings[idx];
            if( !dstr )
                return;
            char* buffer = d_buffer + d_offsets[idx];
            d_results[idx] = dstr->lstrip(d_strip,buffer);
        });
    //
    hipError_t err = hipDeviceSynchronize();
    double et2 = GetTime();
    if( err != hipSuccess )
    {
        fprintf(stderr,"nvs-lstrip(%s)\n",to_strip);
        printCudaError(err);
    }
    pImpl->addOpTimes("lstrip",(et1-st1),(et2-st2));
    if( d_strip )
        RMM_FREE(d_strip,0);
    return rtn;
}

// remove the target character from the beginning and the end of each string
NVStrings* NVStrings::strip( const char* to_strip )
{
    unsigned int count = size();
    custring_view_array d_strings = pImpl->getStringsPtr();
    auto execpol = rmm::exec_policy(0);

    char* d_strip = 0;
    if( to_strip )
    {
        int len = (int)strlen(to_strip) + 1; // include null
        RMM_ALLOC(&d_strip,len,0);
        hipMemcpy(d_strip,to_strip,len,hipMemcpyHostToDevice);
    }

    // compute size of output buffer
    rmm::device_vector<size_t> lengths(count,0);
    size_t* d_lengths = lengths.data().get();
    double st1 = GetTime();
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
        [d_strings, d_strip, d_lengths] __device__(unsigned int idx){
            custring_view* dstr = d_strings[idx];
            if( !dstr )
                return;
            unsigned int len = dstr->strip_size(d_strip);
            len = ALIGN_SIZE(len);
            d_lengths[idx] = (size_t)len;
        });
    // create output object
    NVStrings* rtn = new NVStrings(count);
    char* d_buffer = rtn->pImpl->createMemoryFor(d_lengths);
    if( d_buffer==0 )
    {
        if( d_strip )
            RMM_FREE(d_strip,0);
        return rtn;
    }

    double et1 = GetTime();
    // create offsets
    rmm::device_vector<size_t> offsets(count,0);
    thrust::exclusive_scan(execpol->on(0),lengths.begin(),lengths.end(),offsets.begin());
    // do the strip
    custring_view_array d_results = rtn->pImpl->getStringsPtr();
    size_t* d_offsets = offsets.data().get();
    double st2 = GetTime();
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
        [d_strings, d_strip, d_buffer, d_offsets, d_results] __device__(unsigned int idx){
            custring_view* dstr = d_strings[idx];
            if( !dstr )
                return;
            char* buffer = d_buffer + d_offsets[idx];
            d_results[idx] = dstr->strip(d_strip,buffer);
        });
    //
    hipError_t err = hipDeviceSynchronize();
    double et2 = GetTime();
    if( err != hipSuccess )
    {
        fprintf(stderr,"nvs-strip(%s)\n",to_strip);
        printCudaError(err);
    }
    pImpl->addOpTimes("strip",(et1-st1),(et2-st2));
    if( d_strip )
        RMM_FREE(d_strip,0);
    return rtn;
}

// remove the target character from the end of each string
NVStrings* NVStrings::rstrip( const char* to_strip )
{
    unsigned int count = size();
    custring_view_array d_strings = pImpl->getStringsPtr();
    auto execpol = rmm::exec_policy(0);

    char* d_strip = 0;
    if( to_strip )
    {
        int len = (int)strlen(to_strip) + 1; // include null
        RMM_ALLOC(&d_strip,len,0);
        hipMemcpy(d_strip,to_strip,len,hipMemcpyHostToDevice);
    }

    // compute size of output buffer
    rmm::device_vector<size_t> lengths(count,0);
    size_t* d_lengths = lengths.data().get();
    double st1 = GetTime();
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
        [d_strings, d_strip, d_lengths] __device__(unsigned int idx){
            custring_view* dstr = d_strings[idx];
            if( !dstr )
                return;
            unsigned int len = dstr->rstrip_size(d_strip);
            len = ALIGN_SIZE(len);
            d_lengths[idx] = (size_t)len;
        });

    // create output object
    NVStrings* rtn = new NVStrings(count);
    char* d_buffer = rtn->pImpl->createMemoryFor(d_lengths);
    if( d_buffer==0 )
    {
        if( d_strip )
            RMM_FREE(d_strip,0);
        return rtn; // all strings are null
    }

    double et1 = GetTime();
    // create offsets
    rmm::device_vector<size_t> offsets(count,0);
    thrust::exclusive_scan(execpol->on(0),lengths.begin(),lengths.end(),offsets.begin());
    // do the strip
    custring_view_array d_results = rtn->pImpl->getStringsPtr();
    size_t* d_offsets = offsets.data().get();
    double st2 = GetTime();
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
        [d_strings, d_strip, d_buffer, d_offsets, d_results] __device__(unsigned int idx){
            custring_view* dstr = d_strings[idx];
            if( !dstr )
                return;
            char* buffer = d_buffer + d_offsets[idx];
            d_results[idx] = dstr->rstrip(d_strip,buffer);
        });
    //
    hipError_t err = hipDeviceSynchronize();
    double et2 = GetTime();
    if( err != hipSuccess )
    {
        fprintf(stderr,"nvs-rstrip(%s)\n",to_strip);
        printCudaError(err);
    }
    pImpl->addOpTimes("rstrip",(et1-st1),(et2-st2));
    if( d_strip )
        RMM_FREE(d_strip,0);
    return rtn;
}
